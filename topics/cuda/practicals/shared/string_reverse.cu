#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include "util.hpp"

// TODO : implement a kernel that reverses a string of length n in place
// void reverse_string(char* str, int n)
__global__
void reverse_string_shared(char* str, int n){
    __shared__ char buffer[1024];

    auto idx = threadIdx.x;

    if(idx < n){
        // Load string to buffer
        buffer[idx] = str[idx];

        // Wait for all threads to finish loading
        __syncthreads();

        str[idx] = buffer[n - idx - 1];
    }
}

__global__
void reverse_string_shared_nosync(char* str, int n){
    __shared__ char buffer[1024];
    
    auto idx = threadIdx.x;
    auto idxI = n - threadIdx.x - 1;

    if(idx != idxI && idx < n/2){
        // Load values to swap
        buffer[idx] = str[idx];
        buffer[idxI] = str[idxI];

	// No sync needed (every thread performs an independent swap)

	// Swap
        str[idx] = buffer[idxI];
        str[idxI] = buffer[idx];
    }
}

int main(int argc, char** argv) {
    // check that the user has passed a string to reverse
    if(argc<2) {
        std::cout << "useage : ./string_reverse \"string to reverse\"\n" << std::endl;
        exit(0);
    }

    // determine the length of the string, and copy in to buffer
    auto n = strlen(argv[1]);
    auto string = malloc_managed<char>(n+1);
    std::copy(argv[1], argv[1]+n, string);
    string[n] = 0; // add null terminator

    std::cout << "string to reverse:\n" << string << "\n";

    // TODO : call the string reverse function
    //reverse_string_shared<<<1,n>>>(string, n);    
    //reverse_string_shared<<<1,n>>>(string, n);
    reverse_string_shared_nosync<<<1,n>>>(string, n);

    // print reversed string
    hipDeviceSynchronize();
    std::cout << "reversed string:\n" << string << "\n";

    // free memory
    hipFree(string);

    return 0;
}

